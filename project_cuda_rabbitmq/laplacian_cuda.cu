#include <hip/hip_runtime.h>
#include <stdio.h>

// Standardowy kernel Laplace'a 5x5
__constant__ int laplacian_kernel_5x5[25] = {
    0, 0, -1, 0, 0,
    0, -1, -2, -1, 0,
    -1, -2, 16, -2, -1,
    0, -1, -2, -1, 0,
    0, 0, -1, 0, 0};

// Funkcja pomocnicza do pobierania wartości piksela z obsługą brzegów (clamp to edge)
__device__ unsigned char get_pixel_value(const unsigned char *image, int x, int y, int width, int height, int channel, int channels)
{
    x = max(0, min(x, width - 1));
    y = max(0, min(y, height - 1));
    return image[(y * width + x) * channels + channel];
}

__global__ void apply_laplacian_kernel(
    const unsigned char *input_image,
    unsigned char *output_image,
    int width,
    int height,
    int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        for (int c = 0; c < channels; ++c)
        {
            float sum = 0.0f;
            int kernel_idx = 0;

            for (int ky = -2; ky <= 2; ++ky)
            {
                for (int kx = -2; kx <= 2; ++kx)
                {
                    sum += (float)get_pixel_value(input_image, x + kx, y + ky, width, height, c, channels) *
                           laplacian_kernel_5x5[kernel_idx++];
                }
            }

            // Normalizacja/przycięcie wyniku do zakresu 0-255
            sum = fmaxf(0.0f, fminf(255.0f, sum));
            output_image[(y * width + x) * channels + c] = (unsigned char)sum;
        }
    }
}

extern "C" __declspec(dllexport) void
process_image_cuda(
    const unsigned char *h_input_image,
    unsigned char *h_output_image,
    int width,
    int height,
    int channels,
    float *elapsed_ms)
{
    unsigned char *d_input_image, *d_output_image;
    size_t image_size_bytes = width * height * channels * sizeof(unsigned char);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Alokacja pamięci na GPU
    hipMalloc((void **)&d_input_image, image_size_bytes);
    hipMalloc((void **)&d_output_image, image_size_bytes);

    // Kopiowanie danych z hosta do urządzenia
    hipMemcpy(d_input_image, h_input_image, image_size_bytes, hipMemcpyHostToDevice);

    // Konfiguracja siatki i bloków
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x,
                    (height + threads_per_block.y - 1) / threads_per_block.y);

    // Rozpocznij pomiar czasu
    hipEventRecord(start);

    // Wywołanie kernela
    apply_laplacian_kernel<<<num_blocks, threads_per_block>>>(
        d_input_image, d_output_image, width, height, channels);
    hipDeviceSynchronize();

    // Zatrzymaj pomiar czasu
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsed_ms, start, stop);

    // Kopiowanie wyników z urządzenia do hosta
    hipMemcpy(h_output_image, d_output_image, image_size_bytes, hipMemcpyDeviceToHost);

    // Zwolnienie pamięci na GPU
    hipFree(d_input_image);
    hipFree(d_output_image);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}